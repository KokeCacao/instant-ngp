#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   main.cu
 *  @author Thomas Müller, NVIDIA
 */

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <args/args.hxx>

#include <filesystem/path.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <fcntl.h>
#include <time.h>

using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;
namespace fs = ::filesystem;

/*! Try to get lock. Return its file descriptor or -1 if failed.
 *
 *  @param lockName Name of file used as lock (i.e. '/var/lock/myLock').
 *  @return File descriptor of lock file, or -1 if failed.
 */
int tryGetLock(char const *lockName) {
    mode_t m = umask(0);
    int fd = open(lockName, O_RDWR|O_CREAT, 0666);
    umask(m);
    if ( fd >= 0 && flock(fd, LOCK_EX | LOCK_NB) < 0) {
        close(fd);
        fd = -1;
    }
    return fd;
}

/*! Release the lock obtained with tryGetLock( lockName ).
 *
 *  @param fd File descriptor of lock returned by tryGetLock( lockName ).
 *  @param lockName Name of file used as lock (i.e. '/var/lock/myLock').
 */
void releaseLock(int fd, char const *lockName) {
    if (fd < 0) return;
    close(fd);
}

float standardDevation(std::vector<float> v) {
  float sum = std::accumulate(v.begin(), v.end(), 0.0);
  float mean = sum / v.size();

  std::vector<float> diff(v.size());
  std::transform(v.begin(), v.end(), diff.begin(),
                std::bind2nd(std::minus<float>(), mean));
  float sq_sum = std::inner_product(diff.begin(), diff.end(), diff.begin(), 0.0);
  return std::sqrt(sq_sum / v.size());
}

int main(int argc, char** argv) {
	ArgumentParser parser{
		"neural graphics primitives\n"
		"version " NGP_VERSION,
		"",
	};

	HelpFlag help_flag{
		parser,
		"HELP",
		"Display this help menu.",
		{'h', "help"},
	};

	ValueFlag<string> mode_flag{
		parser,
		"MODE",
		"Mode can be 'nerf', 'sdf', or 'image' or 'volume'. Inferred from the scene if unspecified.",
		{'m', "mode"},
	};

	ValueFlag<string> network_config_flag{
		parser,
		"CONFIG",
		"Path to the network config. Uses the scene's default if unspecified.",
		{'n', 'c', "network", "config"},
	};

	Flag no_gui_flag{
		parser,
		"NO_GUI",
		"Disables the GUI and instead reports training progress on the command line.",
		{"no-gui"},
	};

	Flag no_train_flag{
		parser,
		"NO_TRAIN",
		"Disables training on startup.",
		{"no-train"},
	};

	ValueFlag<string> scene_flag{
		parser,
		"SCENE",
		"The scene to load. Can be NeRF dataset, a *.obj mesh for training a SDF, an image, or a *.nvdb volume.",
		{'s', "scene"},
	};

	ValueFlag<string> lock_flag{
		parser,
		"LOCK",
		"Stream lock.",
		{"lock"},
	};

	ValueFlag<string> change_flag{
		parser,
		"CHANGE",
		"Indicate whether the dataset is changed.",
		{"change"},
	};

	ValueFlag<string> snapshot_flag{
		parser,
		"SNAPSHOT",
		"Optional snapshot to load upon startup.",
		{"snapshot"},
	};

	ValueFlag<uint32_t> width_flag{
		parser,
		"WIDTH",
		"Resolution width of the GUI.",
		{"width"},
	};

	ValueFlag<uint32_t> height_flag{
		parser,
		"HEIGHT",
		"Resolution height of the GUI.",
		{"height"},
	};

	Flag version_flag{
		parser,
		"VERSION",
		"Display the version of neural graphics primitives.",
		{'v', "version"},
	};

	// Parse command line arguments and react to parsing
	// errors using exceptions.
	try {
		parser.ParseCLI(argc, argv);
	} catch (const Help&) {
		cout << parser;
		return 0;
	} catch (const ParseError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -1;
	} catch (const ValidationError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -2;
	}

	if (version_flag) {
		tlog::none() << "neural graphics primitives version " NGP_VERSION;
		return 0;
	}

	try {
		ETestbedMode mode;
		if (!mode_flag) {
			if (!scene_flag) {
				tlog::error() << "Must specify either a mode or a scene";
				return 1;
			}

			fs::path scene_path = get(scene_flag);
			if (!scene_path.exists()) {
				tlog::error() << "Scene path " << scene_path << " does not exist.";
				return 1;
			}

			if (scene_path.is_directory() || equals_case_insensitive(scene_path.extension(), "json")) {
				mode = ETestbedMode::Nerf;
			} else if (equals_case_insensitive(scene_path.extension(), "obj") || equals_case_insensitive(scene_path.extension(), "stl")) {
				mode = ETestbedMode::Sdf;
			} else if (equals_case_insensitive(scene_path.extension(), "nvdb")) {
				mode = ETestbedMode::Volume;
			} else {
				mode = ETestbedMode::Image;
			}
		} else {
			auto mode_str = get(mode_flag);
			if (equals_case_insensitive(mode_str, "nerf")) {
				mode = ETestbedMode::Nerf;
			} else if (equals_case_insensitive(mode_str, "sdf")) {
				mode = ETestbedMode::Sdf;
			} else if (equals_case_insensitive(mode_str, "image")) {
				mode = ETestbedMode::Image;
			} else if (equals_case_insensitive(mode_str, "volume")) {
				mode = ETestbedMode::Volume;
			} else {
				tlog::error() << "Mode must be one of 'nerf', 'sdf', 'image', and 'volume'.";
				return 1;
			}
		}

		Testbed testbed{mode};

		if (scene_flag) {
			fs::path scene_path = get(scene_flag);
			if (!scene_path.exists()) {
				tlog::error() << "Scene path " << scene_path << " does not exist.";
				return 1;
			}
			testbed.load_training_data(scene_path.str());
		}

		std::string mode_str;
		switch (mode) {
			case ETestbedMode::Nerf:   mode_str = "nerf";   break;
			case ETestbedMode::Sdf:    mode_str = "sdf";    break;
			case ETestbedMode::Image:  mode_str = "image";  break;
			case ETestbedMode::Volume: mode_str = "volume"; break;
		}

		if (snapshot_flag) {
			// Load network from a snapshot if one is provided
			fs::path snapshot_path = get(snapshot_flag);
			if (!snapshot_path.exists()) {
				tlog::error() << "Snapshot path " << snapshot_path << " does not exist.";
				return 1;
			}

			testbed.load_snapshot(snapshot_path.str());
			testbed.m_train = false;
		} else {
			// Otherwise, load the network config and prepare for training
			fs::path network_config_path = fs::path{"configs"}/mode_str;
			if (network_config_flag) {
				auto network_config_str = get(network_config_flag);
				if ((network_config_path/network_config_str).exists()) {
					network_config_path = network_config_path/network_config_str;
				} else {
					network_config_path = network_config_str;
				}
			} else {
				network_config_path = network_config_path/"base.json";
			}

			if (!network_config_path.exists()) {
				tlog::error() << "Network config path " << network_config_path << " does not exist.";
				return 1;
			}

			testbed.reload_network_from_file(network_config_path.str());
			testbed.m_train = !no_train_flag;
		}

		bool gui = !no_gui_flag;
#ifndef NGP_GUI
		gui = false;
#endif

		if (gui) {
			testbed.init_window(width_flag ? get(width_flag) : 1920, height_flag ? get(height_flag) : 1080);
		}

		// Render/training loop
    std::cout << "Koke_Cacao: enter training loop" << std::endl;

		while (testbed.frame()) {
			if (!gui) {
				tlog::info() << "iteration=" << testbed.m_training_step << " loss=" << testbed.m_loss_scalar.val();
			}
      std::cout << "\33[2K\rKoke_Cacao: Iteration=" << testbed.m_training_step << " Loss=" << testbed.m_loss_scalar.val() << std::flush;

      // calculate if loss stops decay
      vector<float> subvector_left = {testbed.m_loss_graph.begin(), testbed.m_loss_graph.end() - 1};
      vector<float> subvector_right = {testbed.m_loss_graph.begin() + 1, testbed.m_loss_graph.end()};
      std::transform(subvector_right.begin(), subvector_right.end(), subvector_left.begin(), subvector_right.begin(), std::minus<float>());
      float stdev = standardDevation(subvector_right);
      // tlog::info() << "step: " << testbed.m_training_step << ", sample: " << testbed.m_loss_graph_samples << ", stdev: " << stdev;

      if (lock_flag && scene_flag && testbed.m_loss_graph_samples > testbed.m_loss_graph.size() && stdev < 0.10f) {
        std::cout << "\33[2K\rKoke_Cacao: Iteration=" << testbed.m_training_step << " Loss=" << testbed.m_loss_scalar.val() << " Slow Training!" << std::flush;
        fs::path lock_path = get(lock_flag);
        const std::string& str = lock_path.str();
        const char *cstr = str.c_str();
        int lock_fd = tryGetLock(cstr);
        if (lock_fd > -1) {
          // 1. if this doesn't unlock, blender could not execute (blender can read ngp lock)
          // 2. if I don't load training data here, ngp will wait for blender finish execute (npg can read blender's lock, npg does not access illegal stuff outside of lock)
          // 3. if ngp got lock, and blender change file, it will break
          fs::path scene_path = get(scene_flag);
          const std::string& scene_string = scene_path.str();
          fs::path change_path = get(change_flag);

          if (change_path.exists()) {
            clock_t start = clock();
            change_path.remove_file();
            testbed.load_training_data(scene_string);
            std::cout << std::endl;
            tlog::info() << "Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms. Will Release Lock";
            std::cout << std::endl;
          }

          releaseLock(lock_fd, cstr);
        } else {
          std::cout << std::endl;
				  tlog::warning() << "Cannot Aquire Lock at " << lock_path.str();
          std::cout << std::endl;
        }
      }
		}
	} catch (const exception& e) {
		tlog::error() << "Uncaught exception: " << e.what();
		return 1;
	}
}
